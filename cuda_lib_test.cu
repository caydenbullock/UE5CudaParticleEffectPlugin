#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>

#include "cuda_lib_test.h"
#include <math.h>


__global__ void ParticleKernel(
    float3* positions,
    float3* velocities,
    int numParticles,
    float dt,
    float damping,
    float3 center)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= numParticles) return;

    float3 pos = positions[id];
    float3 vel = velocities[id];

    float3 toCenter = {
        center.x - pos.x,
        center.y - pos.y,
        center.z - pos.z
    };

    float dist2 = toCenter.x * toCenter.x + toCenter.y * toCenter.y + toCenter.z * toCenter.z;
    float dist = sqrtf(dist2);

    if (dist > 1e-4f) 
    {
        toCenter.x /= dist;
        toCenter.y /= dist;
        toCenter.z /= dist;

        float pullStrength = 1500.0f; 
        vel.x += toCenter.x * pullStrength * dt;
        vel.y += toCenter.y * pullStrength * dt;
        vel.z += toCenter.z * pullStrength * dt;
    }

    vel.x *= damping;
    vel.y *= damping;
    vel.z *= damping;

    pos.x += vel.x * dt;
    pos.y += vel.y * dt;
    pos.z += vel.z * dt;

    // Clamp nonsense
    if (!isfinite(pos.x)) pos.x = 0;
    if (!isfinite(pos.y)) pos.y = 0;
    if (!isfinite(pos.z)) pos.z = 0;
    if (!isfinite(vel.x)) vel.x = 0;
    if (!isfinite(vel.y)) vel.y = 0;
    if (!isfinite(vel.z)) vel.z = 0;

    positions[id] = pos;
    velocities[id] = vel;
}

extern "C" void launchParticleKernel(
    float3* positions,
    float3* velocities,
    int numParticles,
    float dt,
    float damping,
    float3 center)
{
    if (numParticles <= 0 || positions == nullptr || velocities == nullptr) return;

    int threadsPerBlock = 256;
    int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;

    ParticleKernel << <blocksPerGrid, threadsPerBlock >> > (
        positions,
        velocities,
        numParticles,
        dt,
        damping,
        center
        );

    hipDeviceSynchronize();
}
